
#include <hip/hip_runtime.h>
///
/// @copyright Copyright (c) 2016-, Issam SAID <said.issam@gmail.com>
/// All rights reserved.
///
/// Redistribution and use in source and binary forms, with or without
/// modification, are permetted provided that the following conditions
/// are met:
///
/// 1. Redistributions of source code must retain the above copyright
///    notice, this list of conditions and the following disclaimer.
/// 2. Redistributions in binary form must reproduce the above copyright
///    notice, this list of conditions and the following disclaimer in the
///    documentation and/or other materials provided with the distribution.
/// 3. Neither the name of the COPYRIGHT HOLDER nor the names of its contributors
///    may be used to endorse or promote products derived from this software
///    without specific prior written permission.
///
/// THIS SOFTWARE IS PROVIDED "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES,
/// INCLUDING, BUT NOT LIMITED TO, WARRANTIES OF MERCHANTABILITY AND FITNESS
/// FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT 
/// HOLDER OR ITS CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, 
/// SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
/// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
/// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF 
/// LIABILITY, WETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
/// NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
/// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
///
/// @file test/src/mem_test.cu
/// @author Issam SAID
/// @brief the CUDA kernels used for ezCU memory utilities tesing.
///
#define S(m, z, y, x) m[(8+dimx)*((8+dimy)*(z+4) + y+4) + x+4]
#define DS      4
#define ZRO     4
#define ONE     5
#define TWO     6
#define TRE     7
#define FOR     8
#define   U0(z, y, x)    u0[(2*DS+dimx)*((2*DS+dimy)*(z+DS) + (y+DS)) + x+DS]
#define   U1(z, y, x)    u1[(2*DS+dimx)*((2*DS+dimy)*(z+DS) + y+DS) + x+DS]
#define ROC2(z, y, x)  roc2[dimx*(dimy*(z) + y ) + x] 
#define  ETA(z, y, x)   eta[(dimx+2)*((dimy+2)*(z+1) + (y+1) ) + (x+1)]
#define  PHI(z, y, x)   phi[dimx*(dimy*(z) + y ) + x]
#define    L(y, x)        l[y+DS][x+DS]

extern "C" __global__ void kselect(float *input, 
                                   int     dimx, 
                                   int     dimy, 
                                   int     dimz) {
    int xgid = threadIdx.x + blockIdx.x*blockDim.x;
    int ygid = threadIdx.y + blockIdx.y*blockDim.y;
    int zgid;
    // Z
    if( (xgid<dimx) && (ygid<dimy) ) { 
        for(zgid = 0; zgid < 4; ++zgid) {
            S(input, zgid, ygid, xgid) = 
                (8+dimx)*((8+dimy)*(zgid+4) + (ygid+4)) + xgid+4;
        }
        for(zgid = dimz-4; zgid < dimz; ++zgid) {
            S(input, zgid, ygid, xgid) = 
                (8+dimx)*((8+dimy)*(zgid+4) + (ygid+4)) + xgid+4;
        }
    }
    // X
    if(( (xgid >= 0) && (xgid<4) ) || ( (xgid >= (dimx-4)) && (xgid<dimx) )) { 
        for(zgid = 0; zgid < dimz; ++zgid) {
            S(input, zgid, ygid, xgid) = 
                (8+dimx)*((8+dimy)*(zgid+4) + (ygid+4)) + xgid+4;
        }
    }
    // Y
    if(( (ygid >= 0) && (ygid<4) ) || ( (ygid >= (dimy-4)) && (ygid<dimy) )) { 
        for(zgid = 0; zgid < dimz; ++zgid) {
            S(input, zgid, ygid, xgid) = 
                (8+dimx)*((8+dimy)*(zgid+4) + (ygid+4)) + xgid+4;
        }
    }

}

///
/// size should be dimx x dimy
/// pack dimx * dimy * lz entries
/// tab[DS][dimy][dimx]
///
extern "C" __global__ void pack_south(int   dimx, 
                                      int   dimy, 
                                      int   dimz,
                                      float  *u0,
                                      float *tab) {
    int xgid = threadIdx.x + blockIdx.x*blockDim.x;
    int ygid = threadIdx.y + blockIdx.y*blockDim.y;
    int zgid;
    
    if ((xgid<dimx) && (ygid<dimy)) { 
        for(zgid = 0; zgid < DS; ++zgid) {
            tab[xgid + dimx*(ygid+(zgid*dimy))] = U0(zgid, ygid, xgid);
        }
    }
}

///
/// size should be dimx x dimy
/// unpack dimx * dimy * lz entries
/// tab[DS][dimy][dimx]
///
extern "C" __global__ void unpack_south(int  dimx, 
                                        int  dimy, 
                                        int  dimz,
                                        float *u0,
                                        float *tab) {
    int xgid = threadIdx.x + blockIdx.x*blockDim.x;
    int ygid = threadIdx.y + blockIdx.y*blockDim.y;
    int zgid;
    
    if ((xgid<dimx) && (ygid<dimy)) { 
        for(zgid = 0; zgid < DS; ++zgid) {
            U0(zgid, ygid, xgid) = tab[xgid + dimx*(ygid+(zgid*dimy))];
        }
    }
}

///
/// size should be dimx x dimy
/// pack dimx * dimy * lz entries
/// tab[DS][dimy][dimx]
///
extern "C" __global__ void pack_north(int  dimx, 
                                      int  dimy, 
                                      int  dimz,
                                      float *u0,
                                      float *tab) {
    int xgid = threadIdx.x + blockIdx.x*blockDim.x;
    int ygid = threadIdx.y + blockIdx.y*blockDim.y;
    int zgid;
    
    if ((xgid<dimx) && (ygid<dimy)) { 
        for(zgid = dimz-DS; zgid < dimz; ++zgid) {
            tab[xgid + dimx*(ygid+((zgid-dimz+DS)*dimy))] = 
                U0(zgid, ygid, xgid);
        }
    }
}

///
/// size should be dimx x dimy
/// unpack dimx * dimy * lz entries
/// tab[DS][dimy][dimx]
///
extern "C" __global__ void unpack_north(int  dimx, 
                                        int  dimy, 
                                        int  dimz,
                                        float *u0,
                                        float *tab) {
    int xgid = threadIdx.x + blockIdx.x*blockDim.x;
    int ygid = threadIdx.y + blockIdx.y*blockDim.y;
    int zgid;
    
    if ((xgid<dimx) && (ygid<dimy)) { 
        for(zgid = dimz-DS; zgid < dimz; ++zgid) {
            U0(zgid, ygid, xgid) = tab[xgid + 
                    dimx*(ygid+((zgid-dimz+DS)*dimy))];
        }
    }
}

///
/// size should be DS x dimy
/// pack DS * dimy * dimz entries
/// tab[dimz][dimy][DS]
extern "C" __global__ void pack_west(int  dimx, 
                                     int  dimy, 
                                     int  dimz,
                                     float *u0,
                                     float *tab) {
    int xgid = threadIdx.x + blockIdx.x*blockDim.x;
    int ygid = threadIdx.y + blockIdx.y*blockDim.y;
    int zgid;
    if((xgid >= 0) && (xgid<DS)) { 
        for(zgid = 0; zgid < dimz; ++zgid) {
            tab[xgid + DS*(ygid+(zgid*dimy))] = U0(zgid, ygid, xgid);
        }
    }
}

///
/// size should be DS x dimy
/// unpack DS * dimy * dimz entries
/// tab[dimz][dimy][DS]
///
extern "C" __global__ void unpack_west(int  dimx, 
                                       int  dimy, 
                                       int  dimz,
                                       float *u0,
                                       float *tab) {
    int xgid = threadIdx.x + blockIdx.x*blockDim.x;
    int ygid = threadIdx.y + blockIdx.y*blockDim.y;
    int zgid;
    if((xgid >= 0) && (xgid<DS)) { 
        for(zgid = 0; zgid < dimz; ++zgid) {
            U0(zgid, ygid, xgid) = tab[xgid + DS*(ygid+(zgid*dimy))];
        }
    }
}

///
/// size should be DS x dimy
/// pack DS * dimy * dimz entries
/// tab[dimz][dimy][DS]
///
extern "C" __global__ void pack_east(int  dimx, 
                                     int  dimy, 
                                     int  dimz,
                                     float *u0,
                                     float *tab) {
    int xgid = threadIdx.x + blockIdx.x*blockDim.x;
    int ygid = threadIdx.y + blockIdx.y*blockDim.y;
    int zgid;
    if((xgid>=0) && (xgid<4)) { 
        for(zgid = 0; zgid < dimz; ++zgid) {
            tab[xgid + DS*(ygid+(zgid*dimy))] = U0(zgid, ygid, xgid+dimx-DS);
        }
    }
}

///
/// size should be DS x dimy
/// unpack DS * dimy * dimz entries
/// tab[dimz][dimy][DS]
///
extern "C" __global__ void unpack_east(int  dimx, 
                                       int  dimy, 
                                       int  dimz,
                                       float *u0,
                                       float *tab) {
    int xgid = threadIdx.x + blockIdx.x*blockDim.x;
    int ygid = threadIdx.y + blockIdx.y*blockDim.y;
    int zgid;
    if((xgid>=0) && (xgid<4)) { 
        for(zgid = 0; zgid < dimz; ++zgid) {
            U0(zgid, ygid, xgid+dimx-DS) = tab[xgid + DS*(ygid+(zgid*dimy))];
        }
    }
}

///
/// size should be dimx x DS
/// pack dimx * ly * dimz entries
/// tab[dimz][DS][dimx]
///
extern "C" __global__ void pack_front(int  dimx, 
                                      int  dimy, 
                                      int  dimz,
                                      float *u0,
                                      float *tab) {
    int xgid = threadIdx.x + blockIdx.x*blockDim.x;
    int ygid = threadIdx.y + blockIdx.y*blockDim.y;
    int zgid;
    if((ygid >= 0) && (ygid<DS)) { 
        for(zgid = 0; zgid < dimz; ++zgid) {
            tab[xgid + dimx*(ygid+(zgid*DS))] = U0(zgid, ygid+dimy-DS, xgid);
        }
    }
}

///
/// size should be dimx x DS
/// unpack dimx * ly * dimz entries
/// tab[dimz][DS][dimx]
///
extern "C" __global__ void unpack_front(int  dimx, 
                                        int  dimy, 
                                        int  dimz,
                                        float *u0,
                                        float *tab) {
    int xgid = threadIdx.x + blockIdx.x*blockDim.x;
    int ygid = threadIdx.y + blockIdx.y*blockDim.y;
    int zgid;
    if((ygid >= 0) && (ygid<DS)) { 
        for(zgid = 0; zgid < dimz; ++zgid) {
            U0(zgid, ygid+dimy-DS, xgid) = tab[xgid + dimx*(ygid+(zgid*DS))];
        }
    }
}

///
/// size should be dimx x DS
/// pack dimx * ly * dimz entries
/// tab[dimz][DS][dimx]
///
extern "C" __global__ void pack_back(int  dimx, 
                                     int  dimy, 
                                     int  dimz,
                                     float *u0,
                                     float *tab) {
    int xgid = threadIdx.x + blockIdx.x*blockDim.x;
    int ygid = threadIdx.y + blockIdx.y*blockDim.y;
    int zgid;
    if((ygid >= 0) && (ygid<DS)) { 
        for(zgid = 0; zgid < dimz; ++zgid) {
            tab[xgid + dimx*(ygid+(zgid*DS))] = U0(zgid, ygid, xgid);
        }
    }
}

///
/// size should be dimx x DS
/// unpack dimx * ly * dimz entries
/// tab[dimz][DS][dimx]
///
extern "C" __global__ void unpack_back(int   dimx, 
                                       int   dimy, 
                                       int   dimz,
                                       float  *u0,
                                       float *tab) {
    int xgid = threadIdx.x + blockIdx.x*blockDim.x;
    int ygid = threadIdx.y + blockIdx.y*blockDim.y;
    int zgid;
    if((ygid >= 0) && (ygid<DS)) { 
        for(zgid = 0; zgid < dimz; ++zgid) {
            U0(zgid, ygid, xgid) = tab[xgid + dimx*(ygid+(zgid*DS))];
        }
    }
}
