
#include <hip/hip_runtime.h>
///
/// @copyright Copyright (c) 2016-, Issam SAID <said.issam@gmail.com>
/// All rights reserved.
///
/// Redistribution and use in source and binary forms, with or without
/// modification, are permetted provided that the following conditions
/// are met:
///
/// 1. Redistributions of source code must retain the above copyright
///    notice, this list of conditions and the following disclaimer.
/// 2. Redistributions in binary form must reproduce the above copyright
///    notice, this list of conditions and the following disclaimer in the
///    documentation and/or other materials provided with the distribution.
/// 3. Neither the name of the COPYRIGHT HOLDER nor the names of its contributors
///    may be used to endorse or promote products derived from this software
///    without specific prior written permission.
///
/// THIS SOFTWARE IS PROVIDED "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES,
/// INCLUDING, BUT NOT LIMITED TO, WARRANTIES OF MERCHANTABILITY AND FITNESS
/// FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT 
/// HOLDER OR ITS CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, 
/// SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
/// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
/// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF 
/// LIABILITY, WETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
/// NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
/// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
///
/// @file test/cuda/knl_test.cu
/// @author Issam SAID
/// @brief the CUDA kernels used for ezcu kernel utilities tesing.
/// 

#define G(m, z, y, x) m[(2*sx+dimx)*((2*sy+dimy)*(z+sz) + y+sy) + x+sx]
#define       W(z, x) u[(2*lx+dimx)*(z+lz) + x+lx]

extern "C" __global__ void test_knl_1(float *input, 
                                      float *output,
                                      int dimx) {
  int gid = threadIdx.x + blockIdx.x*blockDim.x;
  output[gid] = input[gid];
}

extern "C" __global__ void test_knl_2(float *input, 
                                      float *output,
                                      float *coefx,
                                      float *coefy,
                                      float *coefz,
                                      int dimx, int dimy, int dimz) {
  int gid = threadIdx.x + blockIdx.x*blockDim.x;
  output[gid] = input[gid];
}

extern "C" __global__ void test_knl_3(float *input, 
                                      float *output,
                                      int dimx, int dimy, int dimz) {
  int gid = threadIdx.x + blockIdx.x*blockDim.x;
  output[gid] = input[gid];
}

extern "C" __global__ void stencil_v_3d(float  *input,
                                        float *output,
                                        float  *coefx,
                                        float  *coefy,
                                        float  *coefz,
                                        int      dimx, 
                                        int      dimy, 
                                        int      dimz,
                                        int        sx,   
                                        int        sy, 
                                        int        sz) {
    int xgid  = threadIdx.x + blockIdx.x*blockDim.x;
    int ygid  = threadIdx.y + blockIdx.y*blockDim.y;
    int zgid;
    
    /// laplacian operator
    float  laplacian;
    float  current;	
    float b00;
    float b01;
    float b02;
    float b03;
    float f00;
    float f01;
    float f02;
    float f03;
    float  coef = coefx[0] + coefy[0] + coefz[0];
       
    if( (xgid<dimx) && (ygid<dimy) ) { 
        
        b03 = 0.;
        b02 = G(input, -4, ygid, xgid);
        b01 = G(input, -3, ygid, xgid);
        b00 = G(input, -2, ygid, xgid);
        current  = G(input, -1, ygid, xgid);
        f00  = G(input,  0, ygid, xgid);
        f01  = G(input,  1, ygid, xgid);
        f02  = G(input,  2, ygid, xgid);
        f03  = G(input,  3, ygid, xgid);
     
	//#pragma unroll 4
        for(zgid = 0; zgid < dimz; zgid++) {
            b03 = b02;
            b02 = b01;
            b01 = b00;
            b00 = current;
            current  = f00;
            f00  = f01;
            f01  = f02;
            f02  = f03;
            f03  = G(input, (zgid+sz), ygid, xgid);

            laplacian  = coef * current
                + coefx[1] * ( G(input, zgid, ygid, xgid+1) +
                               G(input, zgid, ygid, xgid-1) )
                + coefx[2] * ( G(input, zgid, ygid, xgid+2) +
                               G(input, zgid, ygid, xgid-2) )
                + coefx[3] * ( G(input, zgid, ygid, xgid+3) +
                               G(input, zgid, ygid, xgid-3) )
                + coefx[4] * ( G(input, zgid, ygid, xgid+4) +
                               G(input, zgid, ygid, xgid-4) )

                + coefy[1] * ( G(input, zgid, ygid+1, xgid) +
                               G(input, zgid, ygid-1, xgid) )
                + coefy[2] * ( G(input, zgid, ygid+2, xgid) +
                               G(input, zgid, ygid-2, xgid) )
                + coefy[3] * ( G(input, zgid, ygid+3, xgid) +
                               G(input, zgid, ygid-3, xgid) )
                + coefy[4] * ( G(input, zgid, ygid+4, xgid) +
                               G(input, zgid, ygid-4, xgid) )

                + coefz[1] * ( f00 + b00 )
                + coefz[2] * ( f01 + b01 )
                + coefz[3] * ( f02 + b02 )
                + coefz[4] * ( f03 + b03 );
            G(output, zgid, ygid, xgid) = current + laplacian;
        }
    }
}

extern "C" __global__ void add_source_2d(int ix, int iz,
                                         int ix_p, int iz_p, 
                                         float rxt, float rzt, 
                                         int dimx,                         
                                         int lx, int lz, 
                                         float* source, 
                                         int it, float* u) {
  float src = source[it];
  W(iz, ix)     +=  src*(1.f-rxt)*(1.f-rzt);
  W(iz_p, ix)   +=  src*rzt*(1.f-rxt);
  W(iz,   ix_p) +=  src*(1.f-rzt)*rxt;
  W(iz_p, ix_p) +=  src*rzt*rxt;
}
