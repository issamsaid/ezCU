
#include <hip/hip_runtime.h>
///
/// @copyright Copyright (c) 2016-, Issam SAID <said.issam@gmail.com>
/// All rights reserved.
///
/// Redistribution and use in source and binary forms, with or without
/// modification, are permetted provided that the following conditions
/// are met:
///
/// 1. Redistributions of source code must retain the above copyright
///    notice, this list of conditions and the following disclaimer.
/// 2. Redistributions in binary form must reproduce the above copyright
///    notice, this list of conditions and the following disclaimer in the
///    documentation and/or other materials provided with the distribution.
/// 3. Neither the name of the UPMC nor the names of its contributors
///    may be used to endorse or promote products derived from this software
///    without specific prior written permission.
///
/// THIS SOFTWARE IS PROVIDED "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES,
/// INCLUDING, BUT NOT LIMITED TO, WARRANTIES OF MERCHANTABILITY AND FITNESS
/// FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE UPMC OR
/// ITS CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
/// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
/// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
/// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF 
/// LIABILITY, WETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
/// NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
/// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
///
/// @file sgemm/sgemm.cu
/// @author Issam SAID
/// @brief The CUDA kernel used for the matrix multiplication based on ezcu.
///
extern "C" __global__ void sgemm(float *a,
                                 float *b,
                                 float *c,
                                 unsigned int common_dim) {	
    int k, xgid = threadIdx.x + blockIdx.x*blockDim.x;
    int    ygid = threadIdx.y + blockIdx.y*blockDim.y;	
    int  xgsize = 1024;

    float c_ = c[xgid + ygid*xgsize];
   
    for(k = 0; k < common_dim; ++k)
        c_ += a[k + ygid*common_dim]*b[xgid + k*xgsize];

    c[xgid + ygid*xgsize] = c_;
}
