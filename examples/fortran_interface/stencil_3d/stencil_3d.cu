
#include <hip/hip_runtime.h>
///
/// @copyright Copyright (c) 2016, Issam SAID <said.issam@gmail.com>
/// All rights reserved.
///
/// Redistribution and use in source and binary forms, with or without
/// modification, are permetted provided that the following conditions
/// are met:
///
/// 1. Redistributions of source code must retain the above copyright
///    notice, this list of conditions and the following disclaimer.
/// 2. Redistributions in binary form must reproduce the above copyright
///    notice, this list of conditions and the following disclaimer in the
///    documentation and/or other materials provided with the distribution.
/// 3. Neither the name of the COPYRIGHT HOLDER nor the names of its 
///    contributors may be used to endorse or promote products derived from 
///    this software without specific prior written permission.
///
/// THIS SOFTWARE IS PROVIDED "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES,
/// INCLUDING, BUT NOT LIMITED TO, WARRANTIES OF MERCHANTABILITY AND FITNESS
/// FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT 
/// HOLDER OR ITS CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, 
/// SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
/// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
/// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF 
/// LIABILITY, WETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
/// NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
/// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
///
/// @file stencil_3d/stencil_3d.cu
/// @author Issam SAID
/// @brief the CUDA kernel used for the 3D stencil computations based on ezCU.
///
#define G(m, z, y, x) m[(8+dimx)*((8+dimy)*(z+4) + y+4) + x+4]

extern "C" __global__ void stencil_3d(float  *input, 
                                      float *output,
                                      float *coeffs,
                                      int      dimx, 
                                      int      dimy, 
                                      int      dimz) {
    int xgid  = threadIdx.x + blockIdx.x*blockDim.x;
    int ygid  = threadIdx.y + blockIdx.y*blockDim.y;
    int zgid;
    
    /// laplacian operator
    float laplacian;
    float current;	
    float b00;
    float b01;
    float b02;
    float b03;
    float f00;
    float f01;
    float f02;
    float f03;
    float coef = 3*coeffs[0];

       
    if( (xgid<dimx) && (ygid<dimy) ) { 
        
        b03 = 0.;
        b02 = G(input, -4, ygid, xgid);
        b01 = G(input, -3, ygid, xgid);
        b00 = G(input, -2, ygid, xgid);
        current  = G(input, -1, ygid, xgid);
        f00  = G(input,  0, ygid, xgid);
        f01  = G(input,  1, ygid, xgid);
        f02  = G(input,  2, ygid, xgid);
        f03  = G(input,  3, ygid, xgid);
     
        for(zgid = 0; zgid < dimz; zgid++) {
            b03 = b02;
            b02 = b01;
            b01 = b00;
            b00 = current;
            current  = f00;
            f00  = f01;
            f01  = f02;
            f02  = f03;
            f03  = G(input, (zgid+4), ygid, xgid);

            laplacian  = coef * current
                + coeffs[1] * ( G(input, zgid, ygid, xgid+1) +
                                G(input, zgid, ygid, xgid-1) )
                + coeffs[2] * ( G(input, zgid, ygid, xgid+2) +
                                G(input, zgid, ygid, xgid-2) )
                + coeffs[3] * ( G(input, zgid, ygid, xgid+3) +
                                G(input, zgid, ygid, xgid-3) )
                + coeffs[4] * ( G(input, zgid, ygid, xgid+4) +
                                G(input, zgid, ygid, xgid-4) )

                + coeffs[1] * ( G(input, zgid, ygid+1, xgid) +
                                G(input, zgid, ygid-1, xgid) )
                + coeffs[2] * ( G(input, zgid, ygid+2, xgid) +
                                G(input, zgid, ygid-2, xgid) )
                + coeffs[3] * ( G(input, zgid, ygid+3, xgid) +
                                G(input, zgid, ygid-3, xgid) )
                + coeffs[4] * ( G(input, zgid, ygid+4, xgid) +
                                G(input, zgid, ygid-4, xgid) )

                + coeffs[1] * ( f00 + b00 )
                + coeffs[2] * ( f01 + b01 )
                + coeffs[3] * ( f02 + b02 )
                + coeffs[4] * ( f03 + b03 );
            G(output, zgid, ygid, xgid) = current + laplacian;
        }
    }
}
